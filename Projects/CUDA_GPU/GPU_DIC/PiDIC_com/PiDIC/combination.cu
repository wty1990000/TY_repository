#include "hip/hip_runtime.h"
#include "combination.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include "FFT-CC.h"

#define BLOCK_SIZE 16

__global__ void computeICGN(const float* input_dPXY, const float* input_mdR, const float* input_mdRx, const float* input_mdRy, float m_dNormDeltaP,
							const float* input_mdT, const float* input_mBicubic, const int* input_iU, const int* input_iV, 
							int m_iNumberY, int m_iNumberX, int m_iSubsetH, int m_iSubsetW,	int m_iWidth, int m_iHeight, int m_iSubsetY, int m_iSubsetX, int m_iMaxiteration,
							float* output_dP, int* m_iIterationNum,
							float* m_dSubsetR, float* m_dSubsetT, float* m_dJacobian, float* m_dRDescent, float* m_dHessianXY,float* m_dSubsetAveR, float* m_dSubsetAveT, float* m_dError, float* m_dDP)
/*Input: all the const variables
 Output: deformation P matrix
Strategy: Each thread computes one of the 21*21 POIs
*/
{
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int offset = row*m_iNumberX + col;

	float m_dU, m_dV, m_dUx, m_dUy, m_dVx, m_dVy;
	float m_dDU, m_dDV, m_dDUx, m_dDUy, m_dDVx, m_dDVy;
	float m_dSubAveR,m_dSubNorR, m_dSubAveT,m_dSubNorT;
	float m_dWarp[3][3], m_dHessian[6][6], m_dInvHessian[6][6];
	float m_dNumerator[6];
	float m_dTemp;
	int m_iTemp;
	int m_iIteration;
	float m_dWarpX,	m_dWarpY;
	int m_iTempX,m_iTempY; 
	float m_dTempX, m_dTempY;
	

	if(row<m_iNumberY && col<m_iNumberX){
		m_dU = input_iU[offset];
		m_dV = input_iV[offset];
		m_dUx = 0;
		m_dUy = 0;
		m_dVx = 0;
		m_dVy = 0;

		output_dP[offset*6+0] = m_dU;
		output_dP[offset*6+1] = m_dUx;
		output_dP[offset*6+2] = m_dUy;
		output_dP[offset*6+3] = m_dV;
		output_dP[offset*6+4] = m_dVx;
		output_dP[offset*6+5] = m_dVy;

		// Initialize the warp matrix
		m_dWarp[0][0] = 1 + m_dUx;
		m_dWarp[0][1] = m_dUy;
		m_dWarp[0][2] = m_dU;
		m_dWarp[1][0] = m_dVx;
		m_dWarp[1][1] = 1 + m_dVy;
		m_dWarp[1][2] = m_dV;
		m_dWarp[2][0] = 0;
		m_dWarp[2][1] = 0;
		m_dWarp[2][2] = 1;

		// Initialize the Hessian matrix in subset R
		for (int k = 0; k < 6; k++)
		{
			for (int n = 0; n < 6; n++)
			{
				m_dHessian[k][n] = 0;
			}
		}

		// Initialize Subset R
		m_dSubAveR = 0; // R_m
		m_dSubNorR = 0; // T_m
		// Feed the gray intensity to subset R
		for (int l = 0; l < m_iSubsetH; l++)
		{
			for (int m = 0; m < m_iSubsetW; m++)
			{
				m_dSubsetR[l*m_iSubsetW+m] = input_mdR[int(input_dPXY[offset*2+0] - m_iSubsetY + l)*m_iWidth+int(input_dPXY[offset*2+1] - m_iSubsetX + m)];
				m_dSubAveR += (m_dSubsetR[l*m_iSubsetW+m] / (m_iSubsetH * m_iSubsetW));

				// Evaluate the Jacbian dW/dp at (x, 0);
				m_dJacobian[((l*m_iSubsetW+m)*2+0)*6+0] = 1;
				m_dJacobian[((l*m_iSubsetW+m)*2+0)*6+1] = m - m_iSubsetX;
				m_dJacobian[((l*m_iSubsetW+m)*2+0)*6+2] = l - m_iSubsetY;
				m_dJacobian[((l*m_iSubsetW+m)*2+0)*6+3] = 0;
				m_dJacobian[((l*m_iSubsetW+m)*2+0)*6+4] = 0;
				m_dJacobian[((l*m_iSubsetW+m)*2+0)*6+5] = 0;
				m_dJacobian[((l*m_iSubsetW+m)*2+1)*6+0] = 0;
				m_dJacobian[((l*m_iSubsetW+m)*2+1)*6+1] = 0;
				m_dJacobian[((l*m_iSubsetW+m)*2+1)*6+2] = 0;
				m_dJacobian[((l*m_iSubsetW+m)*2+1)*6+3] = 1;
				m_dJacobian[((l*m_iSubsetW+m)*2+1)*6+4] = m - m_iSubsetX;
				m_dJacobian[((l*m_iSubsetW+m)*2+1)*6+5] = l - m_iSubsetY;

				// Compute the steepest descent image DealtR*dW/dp
				for (int k = 0; k < 6; k++)
				{
					m_dRDescent[(l*m_iSubsetW+m)*6+k] = input_mdR[int(input_dPXY[offset*2+0] - m_iSubsetY + l)*m_iWidth+int(input_dPXY[offset*2+1] - m_iSubsetX + m)] * m_dJacobian[((l*m_iSubsetW+m)*2+0)*6+k] + input_mdRy[int(input_dPXY[offset*2+0] - m_iSubsetY + l)*m_iWidth+int(input_dPXY[offset*2+1] - m_iSubsetX + m)] * m_dJacobian[((l*m_iSubsetW+m)*2+1)*6+k];
				}

				// Compute the Hessian matrix
				for (int k = 0; k < 6; k++)
				{
					for (int n = 0; n < 6; n++)
					{
						m_dHessianXY[((l*m_iSubsetW+m)*6+k)*6+n] = m_dRDescent[(l*m_iSubsetW+m)*6+k] * m_dRDescent[(l*m_iSubsetW+m)*6+n]; // Hessian matrix at each point
						m_dHessian[k][n] += m_dHessianXY[((l*m_iSubsetW+m)*6+k)*6+n]; // sum of Hessian matrix at all the points in subset R
					}
				}
			}		
		}
		__syncthreads();
		for (int l = 0; l < m_iSubsetH; l++)
		{
			for (int m = 0; m < m_iSubsetW; m++)
			{
				m_dSubsetAveR[l*m_iSubsetW+m] = m_dSubsetR[l*m_iSubsetW+m] - m_dSubAveR; // R_i - R_m
				m_dSubNorR += pow(m_dSubsetAveR[l*m_iSubsetW+m], 2);
			}
		}
		__syncthreads();
		m_dSubNorR = sqrt(m_dSubNorR); // sqrt (Sigma(R_i - R_m)^2)

		// Invert the Hessian matrix (Gauss-Jordan algorithm)
		for (int l = 0; l < 6; l++)
		{
			for (int m = 0; m < 6; m++)
			{
				if (l == m)
				{
					m_dInvHessian[l][m] = 1;
				}
				else
				{
					m_dInvHessian[l][m] = 0;
				}
			}
		}
		__syncthreads();
		for (int l = 0; l < 6; l++)
		{
			//Find pivot (maximum lth column element) in the rest (6-l) rows
			m_iTemp = l;
			for (int m = l + 1; m < 6; m++)
			{
				if (m_dHessian[m][l] > m_dHessian[m_iTemp][l])
				{
					m_iTemp = m;
				}
			}
			// Swap the row which has maximum lth column element
			if (m_iTemp != l)
			{
				for (int k = 0; k < 6; k++)
				{
					m_dTemp = m_dHessian[l][k];
					m_dHessian[l][k] = m_dHessian[m_iTemp][k];
					m_dHessian[m_iTemp][k] = m_dTemp;

					m_dTemp = m_dInvHessian[l][k];
					m_dInvHessian[l][k] = m_dInvHessian[m_iTemp][k];
					m_dInvHessian[m_iTemp][k] = m_dTemp;
				}
			}
			__syncthreads();
			// Perform row operation to form required identity matrix out of the Hessian matrix
			for (int m = 0; m < 6; m++)
			{
				m_dTemp = m_dHessian[m][l];
				if (m != l)
				{
					for (int n = 0; n < 6; n++)
					{
						m_dInvHessian[m][n] -= m_dInvHessian[l][n] * m_dTemp / m_dHessian[l][l];
						m_dHessian[m][n] -= m_dHessian[l][n] * m_dTemp / m_dHessian[l][l];
					}
				}
				else
				{
					for (int n = 0; n < 6; n++)
					{
						m_dInvHessian[m][n] /= m_dTemp;
						m_dHessian[m][n] /= m_dTemp;
					}
				}
			}
			__syncthreads();
		}

		// Initialize DeltaP,
		m_dDU = 0;
		m_dDUx = 0;
		m_dDUy = 0;
		m_dDV = 0;
		m_dDVx = 0;
		m_dDVy = 0;

		// Perform interative optimization, with pre-set maximum iteration step
		for (m_iIteration = 0; m_iIteration < m_iMaxiteration; m_iIteration++)
		{
			// Fill warpped image into Subset T
			m_dSubAveT = 0;
			m_dSubNorT = 0;
			for (int l = 0; l < m_iSubsetH; l++)
			{			
				for (int m = 0; m < m_iSubsetW; m++)
				{
					// Calculate the location of warped subset T
					m_dWarpX = input_dPXY[offset*2+1] + m_dWarp[0][0] * (m - m_iSubsetX) + m_dWarp[0][1] * (l - m_iSubsetY) + m_dWarp[0][2];
					m_dWarpY = input_dPXY[offset*2+0] + m_dWarp[1][0] * (m - m_iSubsetX) + m_dWarp[1][1] * (l - m_iSubsetY) + m_dWarp[1][2];
					m_iTempX = int(m_dWarpX);
					m_iTempY = int(m_dWarpY);


					if((m_iTempX >=0) && ( m_iTempY >=0) && (m_iTempX<m_iWidth) && (m_iTempY<m_iHeight)){
						m_dTempX = m_dWarpX - m_iTempX;
						m_dTempY = m_dWarpY - m_iTempY;
						// if it is integer-pixel location, feed the gray intensity of T into the subset T
						if ((m_dTempX == 0) && (m_dTempY == 0))
						{
							m_dSubsetT[l*m_iSubsetW+m] = input_mdT[m_iTempY*m_iSubsetW+m_iTempX];
						}
						else
						{
							// If it is sub-pixel location, estimate the gary intensity using interpolation
							m_dSubsetT[l*m_iSubsetW+m] = 0;
							for (int k = 0; k < 4; k++)
							{
								for (int n = 0; n < 4; n++)
								{
									m_dSubsetT[l*m_iSubsetW+m] += input_mBicubic[((m_iTempY*m_iWidth+m_iTempX)*4+k)*4+n] * pow(m_dTempY, k) * pow(m_dTempX, n); 
								}
							}
						}
						m_dSubAveT += (m_dSubsetT[l*m_iSubsetW+m] / (m_iSubsetH * m_iSubsetW));
					}
					else{
						break;
					}
				}
			}
			for (int l = 0; l < m_iSubsetH; l++)
			{
				for (int m = 0; m < m_iSubsetW; m++)
				{
					m_dSubsetAveT[l*m_iSubsetW+m] = m_dSubsetT[l*m_iSubsetW+m] - m_dSubAveT; // T_i - T_m
					m_dSubNorT += pow(m_dSubsetAveT[l*m_iSubsetW+m], 2);
				}
			}
			m_dSubNorT = sqrt(m_dSubNorT); // sqrt (Sigma(T_i - T_m)^2)
		
			// Compute the error image
			for (int k = 0; k < 6; k++)
			{
				m_dNumerator[k] = 0;
			}
			for (int l = 0; l < m_iSubsetH; l++)
			{
				for (int m = 0; m < m_iSubsetW; m++)
				{
					m_dError[l*m_iSubsetW+m] = (m_dSubNorR / m_dSubNorT) * m_dSubsetAveT[l*m_iSubsetW+m] - m_dSubsetAveR[l*m_iSubsetW+m];
				
					// Compute the numerator
					for (int k = 0; k < 6; k++)
					{
						m_dNumerator[k] += (m_dRDescent[(l*m_iSubsetW+m)*6+k] * m_dError[l*m_iSubsetW+m]);
					}
				}
			}

			// Compute DeltaP
			for (int k = 0; k < 6; k++)
			{
				m_dDP[offset*6+k] = 0;
				for (int n = 0; n < 6; n++)
				{
					m_dDP[offset*6+k] += (m_dInvHessian[k][n] * m_dNumerator[n]);
				}
			}
			m_dDU = m_dDP[offset*6+0];
			m_dDUx = m_dDP[offset*6+1];
			m_dDUy = m_dDP[offset*6+2];
			m_dDV = m_dDP[offset*6+3];
			m_dDVx = m_dDP[offset*6+4];
			m_dDVy = m_dDP[offset*6+5];
		
			// Update the warp
			m_dTemp = (1 + m_dDUx) * (1 + m_dDVy) - m_dDUy * m_dDVx;
			//W(P) <- W(P) o W(DP)^-1
			m_dWarp[0][0] = ((1 + m_dUx) * (1 + m_dDVy) - m_dUy * m_dDVx) / m_dTemp;
			m_dWarp[0][1] = (m_dUy * (1 + m_dDUx) - (1 + m_dUx) * m_dDUy) / m_dTemp;
			m_dWarp[0][2] = m_dU + (m_dUy * (m_dDU * m_dDVx - m_dDV - m_dDV * m_dDUx) - (1 + m_dUx) * (m_dDU * m_dDVy + m_dDU - m_dDUy * m_dDV)) / m_dTemp;
			m_dWarp[1][0] = (m_dVx * (1 + m_dDVy) - (1 + m_dVy) * m_dDVx) / m_dTemp;
			m_dWarp[1][1] = ((1 + m_dVy) * (1 + m_dDUx) - m_dVx * m_dDUy) / m_dTemp;
			m_dWarp[1][2] = m_dV + ((1 + m_dVy) * (m_dDU * m_dDVx - m_dDV - m_dDV * m_dDUx) - m_dVx * (m_dDU * m_dDVy + m_dDU - m_dDUy * m_dDV)) / m_dTemp;
			m_dWarp[2][0] = 0;
			m_dWarp[2][1] = 0;
			m_dWarp[2][2] = 1;


			// Update DeltaP
			output_dP[offset*6+0] = m_dWarp[0][2];
			output_dP[offset*6+1] = m_dWarp[0][0] - 1;
			output_dP[offset*6+2] = m_dWarp[0][1];
			output_dP[offset*6+3] = m_dWarp[1][2];
			output_dP[offset*6+4] = m_dWarp[1][0];
			output_dP[offset*6+5] = m_dWarp[1][1] - 1;

			m_dU = output_dP[offset*6+0];
			m_dUx = output_dP[offset*6+1];
			m_dUy = output_dP[offset*6+2];
			m_dV = output_dP[offset*6+3];
			m_dVx = output_dP[offset*6+4];
			m_dVy =output_dP[offset*6+5];

					//Check if the norm of DeltaP is small enough
			if (sqrt(pow(m_dDP[(row*m_iNumberX+col)*6+0], 2) + pow(m_dDP[(row*m_iNumberX+col)*6+1] * m_iSubsetX, 2) + pow(m_dDP[offset*6+2] * m_iSubsetY, 2) + pow(m_dDP[offset*6+3], 2) + pow(m_dDP[offset*6+4] * m_iSubsetX, 2) + pow(m_dDP[offset*6+5] * m_iSubsetY, 2)) < m_dNormDeltaP)
			{
				break;
			}
		}
		m_iIterationNum[row*m_iNumberX+col] = m_iIteration; // save iteration steps taken at this POI		
	}
}
__global__ void RGradient_kernel(const float *d_InputIMGR, const float *d_InputIMGT, const float* __restrict__ d_InputBiubicMatrix,
								 float *d_OutputIMGR, float *d_OutputIMGT, 
								 float *d_OutputIMGRx, float *d_OutputIMGRy,
								 float *d_OutputIMGTx, float *d_OutputIMGTy, float *d_OutputIMGTxy, float *d_OutputdtBicubic,
								 int width, int height)
{
	//The size of input images
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//Temp arrays
	float d_TaoT[16];
	float d_AlphaT[16];

	//The rows and cols of output matrix.

	if((row < height) && (col < width)){
		d_OutputIMGR[row*width+col]  = d_InputIMGR[(row+1)*(width+2)+col+1];
		d_OutputIMGRx[row*width+col] = 0.5 * (d_InputIMGR[(row+1)*(width+2)+col+2] - d_InputIMGR[(row+1)*(width+2)+col]);
		d_OutputIMGRy[row*width+col] = 0.5 * (d_InputIMGR[(row+2)*(width+2)+col+1] - d_InputIMGR[(row)*(width+2)+col+1]);

		d_OutputIMGT[row*width+col]  = d_InputIMGT[(row+1)*(width+2)+col+1];
		d_OutputIMGTx[row*width+col] = 0.5 * (d_InputIMGT[(row+1)*(width+2)+col+2] -d_InputIMGT[(row+1)*(width+2)+col]);
		d_OutputIMGTy[row*width+col] = 0.5 * (d_InputIMGT[(row+2)*(width+2)+col+1] - d_InputIMGT[(row)*(width+2)+col+1]);
		d_OutputIMGTxy[row*width+col]= 0.25 * (d_InputIMGT[(row+2)*(width+2)+col+2]  - d_InputIMGT[(row)*(width+2)+col+2] -d_InputIMGT[(row+2)*(width+2)+col] + d_InputIMGT[(row)*(width+2)+col]);
	}
	__syncthreads();
		if((row < height-1) && (col < width-1)){
		d_TaoT[0] = d_OutputIMGT[row*(width)+col];
		d_TaoT[1] = d_OutputIMGT[row*(width)+col+1];
		d_TaoT[2] = d_OutputIMGT[(row+1)*(width)+col];
		d_TaoT[3] = d_OutputIMGT[(row+1)*(width)+col+1];
		d_TaoT[4] = d_OutputIMGTx[row*(width)+col];
		d_TaoT[5] = d_OutputIMGTx[row*(width)+col+1];
		d_TaoT[6] = d_OutputIMGTx[(row+1)*(width)+col];
		d_TaoT[7] = d_OutputIMGTx[(row+1)*(width)+col+1];
		d_TaoT[8] = d_OutputIMGTy[row*(width)+col];
		d_TaoT[9] = d_OutputIMGTy[row*(width)+col+1];
		d_TaoT[10] = d_OutputIMGTy[(row+1)*(width)+col];
		d_TaoT[11] = d_OutputIMGTy[(row+1)*(width)+col+1];
		d_TaoT[12] = d_OutputIMGTxy[row*(width)+col];
		d_TaoT[13] = d_OutputIMGTxy[row*(width)+col+1];
		d_TaoT[14] = d_OutputIMGTxy[(row+1)*(width)+col];
		d_TaoT[15] = d_OutputIMGTxy[(row+1)*(width)+col+1];
		for(int k=0; k<16; k++){
			d_AlphaT[k] = 0.0;
			for(int l=0; l<16; l++){
				d_AlphaT[k] += (d_InputBiubicMatrix[k*16+l] * d_TaoT[l]);
			}
		}
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+0] = d_AlphaT[0];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+1] = d_AlphaT[1];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+2] = d_AlphaT[2];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+3] = d_AlphaT[3];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+0] = d_AlphaT[4];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+1] = d_AlphaT[5];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+2] = d_AlphaT[6];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+3] = d_AlphaT[7];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+0] = d_AlphaT[8];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+1] = d_AlphaT[9];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+2] = d_AlphaT[10];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+3] = d_AlphaT[11];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+0] = d_AlphaT[12];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+1] = d_AlphaT[13];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+2] = d_AlphaT[14];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+3] = d_AlphaT[15];
	}
	else if(((row >=height-1)&&(row < height)) && ((col >= width-1)&&(col<width))){
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+3] = 0.0;
	}
	

}

void initialize()
/*Purpose: Initialize GPU with CPU
*/
{
	hipFree(0);
}

void combined_functions(const float* h_InputIMGR, const float* h_InputIMGT, const float* m_dPXY, const int& m_iWidth, const int& m_iHeight, const int& m_iSubsetH, const int& m_iSubsetW, const float& m_dNormDeltaP,
						const int& m_iSubsetX, const int& m_iSubsetY, const int& m_iNumberX, const int& m_iNumberY, const int& m_iFFTSubH, const int& m_iFFTSubW, const int& m_maxIteration,
						int* m_iU, int *m_iV, float* m_dZNCC, float* m_dP, int* m_iIterationNum,
						float& precompute_tme, float& fft_time, float& icgn_time, float& total_time)
{
	//Total timer
	StopWatchWin totalT;
	StopWatchWin precompute;
	StopWatchWin icgn;

	//Variables for precomputation
	float* d_OutputIMGR, *d_OutputIMGT, *d_OutputIMGRx, *d_OutputIMGRy,*d_OutputBicubic;
	float *d_InputIMGR, *d_InputIMGT,*d_InputBiubicMatrix;
	float *d_OutputIMGTx, *d_OutputIMGTy, *d_OutputIMGTxy;
	
	const static float h_InputBicubicMatrix[16*16] = {  
													1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
													0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 ,
													-3, 3, 0, 0, -2, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
													2, -2, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0 ,
													0, 0, 0, 0, 0, 0, 0, 0, -3, 3, 0, 0, -2, -1, 0, 0,
													0, 0, 0, 0, 0, 0, 0, 0, 2, -2, 0, 0, 1, 1, 0, 0 , 
													-3, 0, 3, 0, 0, 0, 0, 0, -2, 0, -1, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, -3, 0, 3, 0, 0, 0, 0, 0, -2, 0, -1, 0,  
													9, -9, -9, 9, 6, 3, -6, -3, 6, -6, 3, -3, 4, 2, 2, 1 , 
													-6, 6, 6, -6, -3, -3, 3, 3, -4, 4, -2, 2, -2, -2, -1, -1, 
													2, 0, -2, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0 , 
													0, 0, 0, 0, 2, 0, -2, 0, 0, 0, 0, 0, 1, 0, 1, 0 , 
													-6, 6, 6, -6, -4, -2, 4, 2, -3, 3, -3, 3, -2, -1, -2, -1,
													4, -4, -4, 4, 2, 2, -2, -2, 2, -2, 2, -2, 1, 1, 1, 1 
												   };
	totalT.start();
	precompute.start();
	(hipMalloc((void**)&d_InputIMGR, (m_iWidth+2)*(m_iHeight+2)*sizeof(float)));
	(hipMalloc((void**)&d_InputIMGT, (m_iWidth+2)*(m_iHeight+2)*sizeof(float)));
	(hipMalloc((void**)&d_InputBiubicMatrix, 16*16*sizeof(float)));
	hipMalloc((void**)&d_OutputIMGR, (m_iWidth*m_iHeight)*sizeof(float));
	hipMalloc((void**)&d_OutputIMGRx, (m_iWidth*m_iHeight)*sizeof(float));
	hipMalloc((void**)&d_OutputIMGRy, (m_iWidth*m_iHeight)*sizeof(float));
	hipMalloc((void**)&d_OutputIMGT, (m_iWidth*m_iHeight)*sizeof(float));
	hipMalloc((void**)&d_OutputBicubic, (m_iWidth*m_iHeight*4*4)*sizeof(float));

	(hipMemcpy(d_InputIMGR,h_InputIMGR,(m_iWidth+2)*(m_iHeight+2)*sizeof(float),hipMemcpyHostToDevice));
	(hipMemcpy(d_InputIMGT,h_InputIMGT,(m_iWidth+2)*(m_iHeight+2)*sizeof(float),hipMemcpyHostToDevice));
	(hipMemcpy(d_InputBiubicMatrix,h_InputBicubicMatrix,16*16*sizeof(float),hipMemcpyHostToDevice));
	
	(hipMalloc((void**)&d_OutputIMGTx, m_iWidth*m_iHeight*sizeof(float)));
	(hipMalloc((void**)&d_OutputIMGTy, m_iWidth*m_iHeight*sizeof(float)));
	(hipMalloc((void**)&d_OutputIMGTxy, m_iWidth*m_iHeight*sizeof(float)));

	dim3 dimB(BLOCK_SIZE,BLOCK_SIZE,1);
	dim3 dimG((m_iWidth-1)/BLOCK_SIZE+1,(m_iHeight-1)/BLOCK_SIZE+1,1);

	RGradient_kernel<<<dimG, dimB>>>(d_InputIMGR,d_InputIMGT,d_InputBiubicMatrix,
								d_OutputIMGR, d_OutputIMGT, 
								 d_OutputIMGRx, d_OutputIMGRy,
								 d_OutputIMGTx, d_OutputIMGTy, d_OutputIMGTxy,d_OutputBicubic,
								 m_iWidth, m_iHeight);

	hipFree(d_OutputIMGTx);
	hipFree(d_OutputIMGTy);
	hipFree(d_OutputIMGTxy);
	hipFree(d_InputIMGR);
	hipFree(d_InputIMGT);
	hipFree(d_InputBiubicMatrix);

	precompute.stop();
	precompute_tme = precompute.getTime();



	//Variables of FFT-CC
	float *hInputm_dR; 
	float *hInputm_dT; 
	hInputm_dR = (float*)malloc(m_iWidth*m_iHeight*sizeof(float));
	hInputm_dT = (float*)malloc(m_iWidth*m_iHeight*sizeof(float));
	hipMemcpy(hInputm_dR, d_OutputIMGR,(m_iWidth*m_iHeight)*sizeof(float),hipMemcpyDeviceToHost);
	hipMemcpy(hInputm_dT, d_OutputIMGT,(m_iWidth*m_iHeight)*sizeof(float),hipMemcpyDeviceToHost);
	//-----Start FFT-CC algorithm------
	FFT_CC(hInputm_dR, hInputm_dT, m_dPXY, m_iNumberY,  m_iNumberX, 
			m_iFFTSubH,  m_iFFTSubW,  m_iWidth, m_iHeight, m_iSubsetY, m_iSubsetX,
			m_dZNCC, m_iU, m_iV,fft_time);




	//Variables of IC-GN
	int* dInput_miU;
	int* dInput_miV;
	float* dInput_mdPXY;
	float* dOutput_mdP;
	int* dOutput_miIterationNum;
	float *dm_dSubsetR, *dm_dSubsetT, *dm_dJacobian, *dm_dRdescent, *dm_dHessianXY, *dm_dSubsetAveR, *dm_dSubsetAveT, *dm_dError, *dm_dDP;

	//Variables for ICGN
	hipMalloc((void**)&dInput_miU, (m_iNumberX*m_iNumberY)*sizeof(int));
	hipMalloc((void**)&dInput_miV, (m_iNumberY*m_iNumberX)*sizeof(int));
	hipMalloc((void**)&dOutput_miIterationNum, (m_iNumberY*m_iNumberX)*sizeof(int));
	hipMalloc((void**)&dInput_mdPXY,(m_iNumberY*m_iNumberX)*2*sizeof(float));
	hipMalloc((void**)&dOutput_mdP, (m_iNumberY*m_iNumberX)*6*sizeof(float));

	hipMemcpy(dInput_miU, m_iU, (m_iNumberX*m_iNumberY*sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(dInput_miV, m_iV, (m_iNumberX*m_iNumberY*sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(dInput_mdPXY, m_dPXY, (m_iNumberY*m_iNumberX)*2*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&dm_dSubsetR, m_iSubsetH*m_iSubsetW*sizeof(float));
	hipMalloc((void**)&dm_dSubsetT, m_iSubsetH*m_iSubsetW*sizeof(float));
	hipMalloc((void**)&dm_dJacobian, m_iSubsetH*m_iSubsetW*2*6*sizeof(float));
	hipMalloc((void**)&dm_dRdescent, m_iSubsetH*m_iSubsetW*6*sizeof(float));
	hipMalloc((void**)&dm_dHessianXY, m_iSubsetH*m_iSubsetW*6*6*sizeof(float));
	hipMalloc((void**)&dm_dSubsetAveR, m_iSubsetH*m_iSubsetW*sizeof(float));
	hipMalloc((void**)&dm_dSubsetAveT, m_iSubsetH*m_iSubsetW*sizeof(float));
	hipMalloc((void**)&dm_dError, m_iSubsetH*m_iSubsetW*sizeof(float));
	hipMalloc((void**)&dm_dDP, m_iNumberX*m_iNumberY*6*sizeof(float));


	dim3 dimGrid((m_iNumberY-1)/16+1, (m_iNumberX-1)/16+1,1);
	dim3 dimBlock(16, 16,1);

	icgn.start();
	computeICGN<<<dimGrid,dimBlock>>>(dInput_mdPXY, d_OutputIMGR, d_OutputIMGRx, d_OutputIMGRy, m_dNormDeltaP,
									  d_OutputIMGT, d_OutputBicubic, dInput_miU, dInput_miV, 
									  m_iNumberY, m_iNumberX, m_iSubsetH, m_iSubsetW, m_iWidth, m_iHeight, m_iSubsetY, m_iSubsetX, m_maxIteration,
									  dOutput_mdP,dOutput_miIterationNum,
									  dm_dSubsetR, dm_dSubsetT, dm_dJacobian,dm_dRdescent,dm_dHessianXY,dm_dSubsetAveR,dm_dSubsetAveT,dm_dError,dm_dDP);

	hipMemcpy(m_dP, dOutput_mdP, (m_iNumberY*m_iNumberX)*6*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(m_iIterationNum, dOutput_miIterationNum, (m_iNumberY*m_iNumberX)*sizeof(int), hipMemcpyDeviceToHost);
	icgn.stop();
	icgn_time = icgn.getTime();

	totalT.stop();
	total_time = totalT.getTime();

	hipFree(dm_dSubsetR);
	hipFree(dm_dSubsetT);
	hipFree(dm_dJacobian);
	hipFree(dm_dRdescent);
	hipFree(dm_dHessianXY);
	hipFree(dm_dSubsetAveR);
	hipFree(dm_dSubsetAveT);
	hipFree(dm_dError);
	hipFree(dm_dDP);

	hipFree(d_OutputIMGR);
	hipFree(d_OutputIMGRx);
	hipFree(d_OutputIMGRy);
	hipFree(d_OutputIMGT);
	hipFree(d_OutputBicubic);
	hipFree(dInput_miU);
	hipFree(dInput_miV);
	hipFree(dInput_mdPXY);
	hipFree(dOutput_mdP);
	hipFree(dOutput_miIterationNum);

	free(hInputm_dR);
	free(hInputm_dT);
}