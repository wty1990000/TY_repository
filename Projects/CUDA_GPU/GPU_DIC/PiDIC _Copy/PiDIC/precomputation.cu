#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime_api.h"
#include "helper_functions.h"


#include <stdio.h>

#include "precomputation.cuh"


__global__ void RGradient_kernel(const double *d_InputIMGR, const double *d_InputIMGT, const double* __restrict__ d_InputBiubicMatrix,
								 double *d_OutputIMGR, double *d_OutputIMGT, 
								 double *d_OutputIMGRx, double *d_OutputIMGRy,
								 double *d_OutputIMGTx, double *d_OutputIMGTy, double *d_OutputIMGTxy, double *d_OutputdtBicubic,
								 int width, int height)
{
	//The size of input images
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//Temp arrays
	double d_TaoT[16];
	double d_AlphaT[16];

	//The rows and cols of output matrix.

	if((row < height) && (col < width)){
		d_OutputIMGR[row*width+col]  = d_InputIMGR[(row+1)*(width+2)+col+1];
		d_OutputIMGRx[row*width+col] = 0.5 * (d_InputIMGR[(row+1)*(width+2)+col+2] - d_InputIMGR[(row+1)*(width+2)+col]);
		d_OutputIMGRy[row*width+col] = 0.5 * (d_InputIMGR[(row+2)*(width+2)+col+1] - d_InputIMGR[(row)*(width+2)+col+1]);

		d_OutputIMGT[row*width+col]  = d_InputIMGT[(row+1)*(width+2)+col+1];
		d_OutputIMGTx[row*width+col] = 0.5 * (d_InputIMGT[(row+1)*(width+2)+col+2] -d_InputIMGT[(row+1)*(width+2)+col]);
		d_OutputIMGTy[row*width+col] = 0.5 * (d_InputIMGT[(row+2)*(width+2)+col+1] - d_InputIMGT[(row)*(width+2)+col+1]);
		d_OutputIMGTxy[row*width+col]= 0.25 * (d_InputIMGT[(row+2)*(width+2)+col+2]  - d_InputIMGT[(row)*(width+2)+col+2] -d_InputIMGT[(row+2)*(width+2)+col] + d_InputIMGT[(row)*(width+2)+col]);
	}
	__syncthreads();
		if((row < height-1) && (col < width-1)){
		d_TaoT[0] = d_OutputIMGT[row*(width)+col];
		d_TaoT[1] = d_OutputIMGT[row*(width)+col+1];
		d_TaoT[2] = d_OutputIMGT[(row+1)*(width)+col];
		d_TaoT[3] = d_OutputIMGT[(row+1)*(width)+col+1];
		d_TaoT[4] = d_OutputIMGTx[row*(width)+col];
		d_TaoT[5] = d_OutputIMGTx[row*(width)+col+1];
		d_TaoT[6] = d_OutputIMGTx[(row+1)*(width)+col];
		d_TaoT[7] = d_OutputIMGTx[(row+1)*(width)+col+1];
		d_TaoT[8] = d_OutputIMGTy[row*(width)+col];
		d_TaoT[9] = d_OutputIMGTy[row*(width)+col+1];
		d_TaoT[10] = d_OutputIMGTy[(row+1)*(width)+col];
		d_TaoT[11] = d_OutputIMGTy[(row+1)*(width)+col+1];
		d_TaoT[12] = d_OutputIMGTxy[row*(width)+col];
		d_TaoT[13] = d_OutputIMGTxy[row*(width)+col+1];
		d_TaoT[14] = d_OutputIMGTxy[(row+1)*(width)+col];
		d_TaoT[15] = d_OutputIMGTxy[(row+1)*(width)+col+1];
		for(int k=0; k<16; k++){
			d_AlphaT[k] = 0.0;
			for(int l=0; l<16; l++){
				d_AlphaT[k] += (d_InputBiubicMatrix[k*16+l] * d_TaoT[l]);
			}
		}
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+0] = d_AlphaT[0];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+1] = d_AlphaT[1];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+2] = d_AlphaT[2];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+3] = d_AlphaT[3];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+0] = d_AlphaT[4];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+1] = d_AlphaT[5];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+2] = d_AlphaT[6];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+3] = d_AlphaT[7];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+0] = d_AlphaT[8];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+1] = d_AlphaT[9];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+2] = d_AlphaT[10];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+3] = d_AlphaT[11];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+0] = d_AlphaT[12];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+1] = d_AlphaT[13];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+2] = d_AlphaT[14];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+3] = d_AlphaT[15];
	}
	else {
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+3] = 0.0;
	}
	

}

void precompute_kernel(const double *h_InputIMGR, const double *h_InputIMGT,
								 double *d_OutputIMGR, double *d_OutputIMGT, 
								 double *d_OutputIMGRx, double *d_OutputIMGRy,
								 double *d_OutputdTBicubic,
								 int width, int height, float& time)
{
	StopWatchWin precompute;
	double *d_InputIMGR, *d_InputIMGT,*d_InputBiubicMatrix;
	double *d_OutputIMGTx, *d_OutputIMGTy, *d_OutputIMGTxy;
	
	const static double h_InputBicubicMatrix[16*16] = {  
													1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
													0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 ,
													-3, 3, 0, 0, -2, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
													2, -2, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0 ,
													0, 0, 0, 0, 0, 0, 0, 0, -3, 3, 0, 0, -2, -1, 0, 0,
													0, 0, 0, 0, 0, 0, 0, 0, 2, -2, 0, 0, 1, 1, 0, 0 , 
													-3, 0, 3, 0, 0, 0, 0, 0, -2, 0, -1, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, -3, 0, 3, 0, 0, 0, 0, 0, -2, 0, -1, 0,  
													9, -9, -9, 9, 6, 3, -6, -3, 6, -6, 3, -3, 4, 2, 2, 1 , 
													-6, 6, 6, -6, -3, -3, 3, 3, -4, 4, -2, 2, -2, -2, -1, -1, 
													2, 0, -2, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0 , 
													0, 0, 0, 0, 2, 0, -2, 0, 0, 0, 0, 0, 1, 0, 1, 0 , 
													-6, 6, 6, -6, -4, -2, 4, 2, -3, 3, -3, 3, -2, -1, -2, -1,
													4, -4, -4, 4, 2, 2, -2, -2, 2, -2, 2, -2, 1, 1, 1, 1 
												   };
	(hipMalloc((void**)&d_InputIMGR, (width+2)*(height+2)*sizeof(double)));
	precompute.start();
	(hipMalloc((void**)&d_InputIMGT, (width+2)*(height+2)*sizeof(double)));
	(hipMalloc((void**)&d_InputBiubicMatrix, 16*16*sizeof(double)));

	(hipMemcpy(d_InputIMGR,h_InputIMGR,(width+2)*(height+2)*sizeof(double),hipMemcpyHostToDevice));
	(hipMemcpy(d_InputIMGT,h_InputIMGT,(width+2)*(height+2)*sizeof(double),hipMemcpyHostToDevice));
	(hipMemcpy(d_InputBiubicMatrix,h_InputBicubicMatrix,16*16*sizeof(double),hipMemcpyHostToDevice));
	
	(hipMalloc((void**)&d_OutputIMGTx, width*height*sizeof(double)));
	(hipMalloc((void**)&d_OutputIMGTy, width*height*sizeof(double)));
	(hipMalloc((void**)&d_OutputIMGTxy, width*height*sizeof(double)));

	dim3 dimB(BLOCK_SIZE,BLOCK_SIZE,1);
	dim3 dimG((width-1)/BLOCK_SIZE+1,(height-1)/BLOCK_SIZE+1,1);

	RGradient_kernel<<<dimG, dimB>>>(d_InputIMGR,d_InputIMGT,d_InputBiubicMatrix,
								d_OutputIMGR, d_OutputIMGT, 
								 d_OutputIMGRx, d_OutputIMGRy,
								 d_OutputIMGTx, d_OutputIMGTy, d_OutputIMGTxy,d_OutputdTBicubic,
								 width, height);

	hipDeviceSynchronize();

	hipFree(d_OutputIMGTx);
	hipFree(d_OutputIMGTy);
	hipFree(d_OutputIMGTxy);
	hipFree(d_InputIMGR);
	hipFree(d_InputIMGT);
	hipFree(d_InputBiubicMatrix);

	precompute.stop();
	time = precompute.getTime();
}

