
#include <cassert>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int main()
{
	int dev_count;
	hipDeviceProp_t prop;
	hipGetDeviceCount( &dev_count);
	for (int i = 0; i < dev_count; i++) {
		hipGetDeviceProperties(&prop, i);
	}
	if (prop.deviceOverlap){
		printf("Device support CUDA streams\n");
	}
	printf("Device has %d SMs\n",prop.multiProcessorCount);
	printf("Device has %d threads per SMs",prop.maxThreadsPerMultiProcessor);
	printf("Device has %d threads per block",prop.maxThreadsPerBlock);
	
	return 0;
}