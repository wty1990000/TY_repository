#include "hip/hip_runtime.h"
#include "CudaComputing.cuh"

#include "hip/hip_runtime.h"

#include ""

 

hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel(int *c, const int *a, const int *b)

{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
int vvmain(int* c)
{
	const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5 };
    const int b[arraySize] = {10, 20, 30, 40, 50 };
    // Add vectors inparallel.

    hipError_t cudaStatus =addWithCuda(c, a, b, arraySize);
    if (cudaStatus !=hipSuccess) {
        return 1;
    }

 

    // hipDeviceReset must becalled before exiting in order for profiling and

    // tracing tools such asNsight and Visual Profiler to show complete traces.

    cudaStatus =hipDeviceReset();
    if (cudaStatus !=hipSuccess) {
        return 1;
    }
    return 0;
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)
{

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipError_t cudaStatus;
    // Choose which GPU to runon, change this on a multi-GPU system
    cudaStatus =hipSetDevice(0);
    if (cudaStatus !=hipSuccess) {
       goto Error;
    }
    // Allocate GPU buffersfor three vectors (two input, one output)   .
    cudaStatus =hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus !=hipSuccess) {
        goto Error;
    }
    cudaStatus =hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus !=hipSuccess) {
        goto Error;
    }
    cudaStatus =hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus !=hipSuccess) {
        goto Error;

    }
    // Copy input vectors fromhost memory to GPU buffers.
    cudaStatus =hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus !=hipSuccess) {
        goto Error;
    }
    cudaStatus =hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus !=hipSuccess) {
        goto Error;
    }
    // Launch a kernel on theGPU with one thread for each element.
    addKernel<<<(size-1)/128+1,128>>>(dev_c, dev_a, dev_b);
    // cudaDeviceSynchronizewaits for the kernel to finish, and returns
    // any errors encounteredduring the launch.
    cudaStatus =hipDeviceSynchronize();
    if (cudaStatus !=hipSuccess) {
        goto Error;
    }
    // Copy output vector fromGPU buffer to host memory.
    cudaStatus = hipMemcpy(c,dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus !=hipSuccess) {
        goto Error;
    }
Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return cudaStatus;
}