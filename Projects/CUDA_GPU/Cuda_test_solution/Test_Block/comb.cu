#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "comb.cuh"
#include "kernel.cuh"


void combine(float* output, const int& sizeo, const int& sizei)
{
	float *dOutput;
	checkCudaErrors(hipMalloc((void**)&dOutput, sizeo*sizeo*sizeof(float)));

	launch_kernel(dOutput, sizeo, sizei);

	checkCudaErrors(hipMemcpy(output, dOutput, sizeo*sizeo*sizeof(float),hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(dOutput));
}