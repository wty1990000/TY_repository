#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

__global__ void assignment(float* output)
{
	unsigned int blockID = blockIdx.y * gridDim.x + blockIdx.x;

	if(threadIdx.x ==0 && threadIdx.y == 0){
		output[blockID] = 20.0;
	}
}

void launch_kernel(float *dOutput, const int& sizeo, const int& sizei)
{
	dim3 dimG(sizeo, sizeo, 1);
	dim3 dimB(sizei, sizei, 1);

	assignment<<<dimG,dimB>>>(dOutput);
}

