#include "hip/hip_runtime.h"
#include    <wb.h>

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
			             int numARows, int numAColumns,
			             int numBRows, int numBColumns,
			             int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    //@@ You have to use shared memory for this MP
	__shared__ float ds_A[16][16];
	__shared__ float ds_B[16][16];
	
	int ty = threadIdx.y;  int tx = threadIdx.x;
	
	int Row = ty + blockDim.y*blockIdx.y;
	int Col = tx + blockDim.x*blockIdx.x;
	float CValue = 0.0;
	
	for(int i=0; i < (numAColumns-1)/16+1; ++i){
		if((Row<numARows) && (i*16+tx < numAColumns))
		{ds_A[ty][tx] = A[numAColumns*Row + i*16+tx];}
		else{
			ds_A[ty][tx] = 0.0;}
		if((i*16+ty < numBRows) && (Col<numBColumns)){
			ds_B[ty][tx] = B[(i*16+ty)*numBColumns+Col];}
		else{
			ds_B[ty][tx] = 0.0;}
		__syncthreads();
		for(int j=0; j<16; ++j){
			CValue += ds_A[ty][j] * ds_B[j][tx];}
	__syncthreads();	
	}
	if((Row<numCRows) && (Col<numCColumns)){
		C[Row*numCColumns+Col] = CValue;}
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    //@@ Allocate the hostC matrix
	hostC = (float *) malloc(numCRows*numCColumns*sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
	wbCheck(hipMalloc((void**)&deviceA, numARows*numAColumns*sizeof(float)));
	wbCheck(hipMalloc((void**)&deviceB, numBRows*numBColumns*sizeof(float)));
	wbCheck(hipMalloc((void**)&deviceC, numCRows*numCColumns*sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
	wbCheck(hipMemcpy(deviceA,hostA,numARows*numAColumns*sizeof(float),hipMemcpyHostToDevice));
	wbCheck(hipMemcpy(deviceB,hostB,numBRows*numBColumns*sizeof(float),hipMemcpyHostToDevice));

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid((numBColumns-1)/16+1, (numARows-1)/16+1,1);
	dim3 DimBlock(16,16,1);
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
	matrixMultiplyShared<<<DimGrid,DimBlock>>>(deviceA, deviceB, deviceC,
                   numARows, numAColumns,
                   numBRows, numBColumns,
                   numCRows, numCColumns);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
	wbCheck(hipMemcpy(hostC,deviceC,numCRows*numCColumns*sizeof(float),hipMemcpyDeviceToHost));

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
	wbCheck(hipFree(deviceA));
	wbCheck(hipFree(deviceB));
	wbCheck(hipFree(deviceC));
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}