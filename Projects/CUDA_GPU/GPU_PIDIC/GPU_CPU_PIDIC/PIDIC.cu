#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "thrust\host_vector.h"
#include "thrust\device_vector.h"
#include "thrust\reduce.h"
#include "EasyBMP.h"
#include "PIDIC.cuh"
#include "FFTCC.h"

#include <stdio.h>
#include <iostream>

//Parameters
const int iMarginX = 10,	iMarginY = 10;
const int iGridX = 10,		iGridY = 10;
const int iSubsetX = 8,	iSubsetY =8;
const float fDeltaP = 0.001f;
const int iIterationNum = 5;
const int BLOCK_SIZE = 16;

//CUDA RUNTIME Initialization
void InitCuda()
{
	hipFree(0);
}

__global__ void precomputation_kernel(float *d_InputIMGR, float *d_InputIMGT, const float* __restrict__ d_InputBiubicMatrix,
								 float *d_OutputIMGR, float *d_OutputIMGT, float *d_OutputIMGRx, float *d_OutputIMGRy,
								 float *d_OutputIMGTx, float *d_OutputIMGTy, float *d_OutputIMGTxy, float *d_OutputdtBicubic,
								 int width, int height)
{
	//The size of input images
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//Temp arrays
	float d_TaoT[16];
	float d_AlphaT[16];

	//The rows and cols of output matrix.

	if((row < height) && (col < width)){
		d_OutputIMGR[row*width+col]  = d_InputIMGR[(row+1)*(width+2)+col+1];
		d_OutputIMGRx[row*width+col] = 0.5 * (d_InputIMGR[(row+1)*(width+2)+col+2] - d_InputIMGR[(row+1)*(width+2)+col]);
		d_OutputIMGRy[row*width+col] = 0.5 * (d_InputIMGR[(row+2)*(width+2)+col+1] - d_InputIMGR[(row)*(width+2)+col+1]);

		d_OutputIMGT[row*width+col]  = d_InputIMGT[(row+1)*(width+2)+col+1];
		d_OutputIMGTx[row*width+col] = 0.5 * (d_InputIMGT[(row+1)*(width+2)+col+2] -d_InputIMGT[(row+1)*(width+2)+col]);
		d_OutputIMGTy[row*width+col] = 0.5 * (d_InputIMGT[(row+2)*(width+2)+col+1] - d_InputIMGT[(row)*(width+2)+col+1]);
		d_OutputIMGTxy[row*width+col]= 0.25 * (d_InputIMGT[(row+2)*(width+2)+col+2]  - d_InputIMGT[(row)*(width+2)+col+2] -d_InputIMGT[(row+2)*(width+2)+col] + d_InputIMGT[(row)*(width+2)+col]);
	}
	__syncthreads();
	if((row < height-1) && (col < width-1)){
		d_TaoT[0] = d_OutputIMGT[row*(width)+col];
		d_TaoT[1] = d_OutputIMGT[row*(width)+col+1];
		d_TaoT[2] = d_OutputIMGT[(row+1)*(width)+col];
		d_TaoT[3] = d_OutputIMGT[(row+1)*(width)+col+1];
		d_TaoT[4] = d_OutputIMGTx[row*(width)+col];
		d_TaoT[5] = d_OutputIMGTx[row*(width)+col+1];
		d_TaoT[6] = d_OutputIMGTx[(row+1)*(width)+col];
		d_TaoT[7] = d_OutputIMGTx[(row+1)*(width)+col+1];
		d_TaoT[8] = d_OutputIMGTy[row*(width)+col];
		d_TaoT[9] = d_OutputIMGTy[row*(width)+col+1];
		d_TaoT[10] = d_OutputIMGTy[(row+1)*(width)+col];
		d_TaoT[11] = d_OutputIMGTy[(row+1)*(width)+col+1];
		d_TaoT[12] = d_OutputIMGTxy[row*(width)+col];
		d_TaoT[13] = d_OutputIMGTxy[row*(width)+col+1];
		d_TaoT[14] = d_OutputIMGTxy[(row+1)*(width)+col];
		d_TaoT[15] = d_OutputIMGTxy[(row+1)*(width)+col+1];
		for(int k=0; k<16; k++){
			d_AlphaT[k] = 0.0;
			for(int l=0; l<16; l++){
				d_AlphaT[k] += (d_InputBiubicMatrix[k*16+l] * d_TaoT[l]);
			}
		}
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+0] = d_AlphaT[0];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+1] = d_AlphaT[1];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+2] = d_AlphaT[2];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+3] = d_AlphaT[3];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+0] = d_AlphaT[4];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+1] = d_AlphaT[5];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+2] = d_AlphaT[6];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+3] = d_AlphaT[7];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+0] = d_AlphaT[8];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+1] = d_AlphaT[9];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+2] = d_AlphaT[10];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+3] = d_AlphaT[11];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+0] = d_AlphaT[12];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+1] = d_AlphaT[13];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+2] = d_AlphaT[14];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+3] = d_AlphaT[15];
	}
	else if(((row >=height-1)&&(row < height)) && ((col >= width-1)&&(col<width))){
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+3] = 0.0;
	}

}
__global__ void ICGN_kernel(float* input_R, float* input_Rx, float* input_Ry, float* input_AveR, float* input_NormR, float fDeltaP,
							float *input_T,   float* input_Bicubic, int* input_iU, int* input_iV, 
							int iNumberY, int iNumberX, int iSubsetH, int iSubsetW, int width, int height, int iSubsetY, int iSubsetX, 
							int iGridSpaceX, int iGridSpaceY, int iMarginX, int iMarginY, int iIterationNum,
							float* output_dP)
/*BLOCK_SIZE: 2*(iSubsetW+1)+1, 2*(iSubsetH+1)+1
  Grid_SIZE:  iNumberX * iNumberY 
*/
{
	int x = threadIdx.x, y = threadIdx.y;
	int offset = blockIdx.y * gridDim.x + blockIdx.x;

	//Shared variables of each ROI
	__shared__ float fPXY[2];
	__shared__ float fT[19*19];
	__shared__ float fBicubic[19*19*4*4];
	__shared__ float fR[17*17];
	__shared__ float fRx[17*17];
	__shared__ float fRy[17*17];
	__shared__ float fdP[6], fdDP[6], fdWarp[3][3], fHessian[6][6],fInvHessian[6][6],fNumerator[6];
	__shared__ float fdU, fdV, fdUx, fdUy, fdVx, fdVy;
	__shared__ float fdDU, fdDUx, fdDUy, fdDV, fdDVx, fdDVy;
	__shared__ float fSubAveR, fSubNormR, fSubAveT, fSubNormT;

	//Private variables for each subset point
	float fJacobian[2][6], fRDescent[6], fHessianXY[6][6];
	float fSubsetR, fSubsetAveR, fSubsetT, fSubsetAveT;
	float fdError;
	
	//Load the auxiliary variables into shared memory of each block
	if(x==0 && y ==0){
		fPXY[0] = float(iMarginX + iSubsetY + blockIdx.y * iGridSpaceY);
		fPXY[1] = float(iMarginY + iSubsetX + blockIdx.x * iGridSpaceX);
		
		fdU = float(input_iU[offset]);		fdDU = 0.0f;
		fdV = float(input_iV[offset]);		fdDV = 0.0f;
		fdUx = 0.0f;						fdDUx = 0.0f;
		fdUy = 0.0f;						fdDUy = 0.0f;
		fdVx = 0.0f;						fdDVx = 0.0f;
		fdVy = 0.0f;						fdDVy = 0.0f;

		fdP[0] = fdU;		fdP[3] = fdV;
		fdP[1] = fdUx;		fdP[4] = fdVx;
		fdP[2] = fdUy;		fdP[5] = fdVy;
	
		fdP[0] = 0.0f;		fdP[3] = 0.0f;
		fdP[1] = 0.0f;		fdP[4] = 0.0f;
		fdP[2] = 0.0f;		fdP[5] = 0.0f;

		fdWarp[0][0] = 1 + fdUx;	fdWarp[0][1] = fdUy;		fdWarp[0][2] = fdU;		
		fdWarp[1][0] = fdVx;		fdWarp[1][1] = 1 + fdVy;	fdWarp[1][2] = fdV;
		fdWarp[2][0] = 0.0f;		fdWarp[2][1] = 0.0f;		fdWarp[2][2] = 1.0f;

		fNumerator[0] = 0.0f;	fNumerator[1] = 0.0f;	fNumerator[2] = 0.0f;	fNumerator[3] = 0.0f;	fNumerator[4] = 0.0f;	fNumerator[5] = 0.0f;
		fdDP[0] = 0.0f;	fdDP[1] = 0.0f;	fdDP[2] = 0.0f;	fdDP[3] = 0.0f;	fdDP[4] = 0.0f;	fdDP[5] = 0.0f;

		fSubAveR = input_AveR[offset];	fSubNormR = input_NormR[offset];
		fSubsetAveT = 0.0f;				fSubNormT = 0.0f;
	}
	__syncthreads();
	if( x<6 && y<6){
		if( x == y){
			fInvHessian[y][x] = 1.0f;
			fHessian[y][x]  = 0.0f;
		}
		else{
			fInvHessian[y][x] = 0.0f;
			fHessian[y][x] = 0.0f;
		}
	}
	__syncthreads();
	
	//Load PXY, Rx, Ry and R, T, Bicubic  into shared_memory
	if( x<iSubsetW && y<iSubsetH ){
		fR[y*iSubsetW+x] = input_R[int()*width+int()];	
	}


	// Evaluate the Jacbian dW/dp at (x, 0);
	fJacobian[0][0] = 1;
	fJacobian[0][1] = x - iSubsetX;
	fJacobian[0][2] = y - iSubsetY;
	fJacobian[0][3] = 0;
	fJacobian[0][4] = 0;
	fJacobian[0][5] = 0;
	fJacobian[1][0] = 0;
	fJacobian[1][1] = 0;
	fJacobian[1][2] = 0;
	fJacobian[1][3] = 1;
	fJacobian[1][4] = x - iSubsetX;
	fJacobian[1][5] = y - iSubsetY;

	for(unsigned int i=0; i<6; i++){
		fRDescent[i] = 
	}

}


void computation_interface(const std::vector<float>& ImgR, const std::vector<float>& ImgT, int iWidth, int iHeight)
{
	//Timers
	StopWatchWin WatchPrecompute, WatchICGN, WatchTotal;
	float fTimePrecopmute=0.0f, fTimeFFTCC=0.0f, fTimeICGN=0.0f, fTimeTotal=0.0f;

	
	
	//Parameters used in the computations.
	int width =  iWidth - 2;
	int height = iHeight -2;
	int iNumberX = int(floor((width - iSubsetX*2 - iMarginX*2)/float(iGridX))) + 1;
	int iNumberY = int(floor((height - iSubsetY*2 - iMarginY*2)/float(iGridY))) + 1;
	int iSubsetW = iSubsetX*2+1;
	int iSubsetH = iSubsetY*2+1;
	int iFFTSubW = iSubsetX*2;
	int iFFTSubH = iSubsetY*2;

	/*--------------------------------------Parameters for CUDA kernel use---------------------------------------------*/
	//Precomputation Parameters
	const static float h_InputBicubicCoeff[16*16] = {  
													1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
													0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 ,
													-3, 3, 0, 0, -2, -1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
													2, -2, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0 ,
													0, 0, 0, 0, 0, 0, 0, 0, -3, 3, 0, 0, -2, -1, 0, 0,
													0, 0, 0, 0, 0, 0, 0, 0, 2, -2, 0, 0, 1, 1, 0, 0 , 
													-3, 0, 3, 0, 0, 0, 0, 0, -2, 0, -1, 0, 0, 0, 0, 0, 
													0, 0, 0, 0, -3, 0, 3, 0, 0, 0, 0, 0, -2, 0, -1, 0,  
													9, -9, -9, 9, 6, 3, -6, -3, 6, -6, 3, -3, 4, 2, 2, 1 , 
													-6, 6, 6, -6, -3, -3, 3, 3, -4, 4, -2, 2, -2, -2, -1, -1, 
													2, 0, -2, 0, 0, 0, 0, 0, 1, 0, 1, 0, 0, 0, 0, 0 , 
													0, 0, 0, 0, 2, 0, -2, 0, 0, 0, 0, 0, 1, 0, 1, 0 , 
													-6, 6, 6, -6, -4, -2, 4, 2, -3, 3, -3, 3, -2, -1, -2, -1,
													4, -4, -4, 4, 2, 2, -2, -2, 2, -2, 2, -2, 1, 1, 1, 1 
												   };
	float *d_InputIMGR, *d_InputIMGT,*d_InputBiubicCoeff;
	float *d_OutputIMGTx, *d_OutputIMGTy, *d_OutputIMGTxy,*d_OutputIMGR, *d_OutputIMGT, *d_OutputIMGRx, *d_OutputIMGRy, *d_OutputBicubic;
	//FFT-ZNCC Parameters
	float *hInput_dR, *hInput_dT, *fZNCC;
	int *iU, *iV;
	
	/*------------------------------Real computation starts here--------------------------------
	  Totally, there are three steps:
	  1. Precomputation of images' gradients matrix and bicubic interpolation matrix
	  2. Using FFT to transform the two images into frequency domain, and after per-
	  forming ZNCC, transforming the results back.
	  3. A Gaussian Newton's optimization method is used to estimate the warped images.
	*/
	WatchTotal.start();

	//Precomputation Starts;
	WatchPrecompute.start();
	checkCudaErrors(hipMalloc((void**)&d_InputIMGR, (width+2)*(height+2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_InputIMGT, (width+2)*(height+2)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_InputBiubicCoeff, 16*16*sizeof(float)));
	checkCudaErrors(hipMemcpy(d_InputIMGR,&ImgR[0],(width+2)*(height+2)*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_InputIMGT,&ImgT[0],(width+2)*(height+2)*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_InputBiubicCoeff,h_InputBicubicCoeff,16*16*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&d_OutputIMGR, (width*height)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_OutputIMGRx, (width*height)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_OutputIMGRy, (width*height)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_OutputIMGT, (width*height)*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_OutputIMGTx, width*height*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_OutputIMGTy, width*height*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_OutputIMGTxy, width*height*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_OutputBicubic, (width*height*4*4)*sizeof(float)));
	dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE,1);
	dim3 dimGirds((width-1)/BLOCK_SIZE+1,(height-1)/BLOCK_SIZE+1,1);
	precomputation_kernel<<<dimGirds,dimBlock>>>(d_InputIMGR,d_InputIMGT,d_InputBiubicCoeff,
						  d_OutputIMGR,d_OutputIMGT,d_OutputIMGRx,d_OutputIMGRy,
						  d_OutputIMGTx,d_OutputIMGTy,d_OutputIMGTxy,d_OutputBicubic,
						  width,height);
	hipFree(d_OutputIMGTx);
	hipFree(d_OutputIMGTy);
	hipFree(d_OutputIMGTxy);
	hipFree(d_InputIMGR);
	hipFree(d_InputIMGT);
	hipFree(d_InputBiubicCoeff);
	WatchPrecompute.stop();
	fTimePrecopmute = WatchPrecompute.getTime();

	//FFT-ZNCC Begins
	hInput_dR = (float*)malloc(width*height*sizeof(float));
	hInput_dT = (float*)malloc(width*height*sizeof(float));
	fZNCC = (float*)malloc(iNumberX*iNumberY*sizeof(float));
	iU = (int*)malloc(iNumberX*iNumberY*sizeof(int));
	iV = (int*)malloc(iNumberX*iNumberY*sizeof(int));
	float *fdPXY = (float*)malloc(iNumberX*iNumberY*2*sizeof(float));
	for(int i=0; i<iNumberY; i++){
		for(int j=0; j<iNumberX; j++){
			fdPXY[(i*iNumberX+j)*2+0] = float(iMarginX + iSubsetY + i*iGridY);
			fdPXY[(i*iNumberX+j)*2+1] = float(iMarginY + iSubsetX + j*iGridX);
		}
	}
	checkCudaErrors(hipMemcpy(hInput_dR, d_OutputIMGR, width*height*sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hInput_dT, d_OutputIMGT, width*height*sizeof(float), hipMemcpyDeviceToHost));
	FFT_CC_interface(hInput_dR, hInput_dT, fdPXY, iNumberY, iNumberX, iFFTSubH, iFFTSubW, 
		width, height, iSubsetY, iSubsetX, fZNCC, iU, iV, fTimeFFTCC);

	WatchTotal.stop();
	fTimeTotal = WatchTotal.getTime();


	/*Befor ICGN starts, pass the average and norm value of image R in first, to simplify GPU's work.*/
	thrust::host_vector<float> hAveR;
	thrust::host_vector<float> hNormR;

	float temp = 0.0f, temp1=0.0f, temp2 = 0.0f;
	for(int i=0; i<iNumberY; i++){
		for(int j=0; j<iNumberX; j++){
			temp = 0.0;
			for(int l=0; l<iSubsetH; l++){
				for(int m=0; m<iSubsetW; m++){
					temp += hInput_dR[int(fdPXY[(i*iNumberX+j)*2+0] - iSubsetY+l)*width
									+ int(fdPXY[(i*iNumberX+j)*2+1] - iSubsetX+m)] / float(iSubsetW*iSubsetH);
				}
			}
			hAveR.push_back(temp);
			temp1 = 0.0f, temp2 = 0.0f;
			for(int l=0; l<iSubsetH; l++){
				for(int m=0; m<iSubsetW; m++){
					temp1 = hInput_dR[int(fdPXY[(i*iNumberX+j)*2+0] - iSubsetY+l)*width
									+ int(fdPXY[(i*iNumberX+j)*2+1] - iSubsetX+m)] / float(iSubsetW*iSubsetH) - hAveR[i*iNumberX+j];
					temp2 += pow(temp1,2);
				}
			}
			hNormR.push_back(sqrt(temp2));
		}
	}
	thrust::device_vector<float> dAveR = hAveR;
	thrust::device_vector<float> dNormR = hNormR;
	float *dAveRaw = thrust::raw_pointer_cast(&dAveR[0]);
	float *dNormRaw = thrust::raw_pointer_cast(&dNormR[0]);

	//ICGN-Begins
	WatchICGN.start();
	int *dInput_iU, *dInput_iV;
	float *dInput_fPXY, *dOutput_fDP;
	checkCudaErrors(hipMalloc((void**)&dInput_iU, (iNumberX*iNumberY)*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&dInput_iV, (iNumberX*iNumberY)*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&dInput_fPXY, (iNumberX*iNumberY)*2*sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dOutput_fDP, (iNumberX*iNumberY)*6*sizeof(float)));
	checkCudaErrors(hipMemcpy(dInput_iU, iU,(iNumberX*iNumberY)*sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dInput_iV, iV,(iNumberX*iNumberY)*sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dInput_fPXY, fdPXY,(iNumberX*iNumberY)*2*sizeof(float), hipMemcpyHostToDevice));

	dim3 dimB((iSubsetW+2),(iSubsetH+2),1);
	dim3 dimG(iNumberX,iNumberY,1);

	ICGN_kernel<<<dimG,dimB>>>(dInput_fPXY,d_OutputIMGR,d_OutputIMGRx,d_OutputIMGRy,fDeltaP,d_OutputIMGT,d_OutputBicubic,dInput_iU,dInput_iV,
		iNumberY,iNumberX,iSubsetH,iSubsetW,width,height,iSubsetY,iSubsetX,iIterationNum,dOutput_fDP);
	float *fdP = (float*)malloc(iNumberX*iNumberY*6*sizeof(float));
	checkCudaErrors(hipMemcpy(fdP, dOutput_fDP, iNumberY*iNumberX*6*sizeof(float), hipMemcpyDeviceToHost));
	WatchICGN.stop();
	fTimeICGN = WatchICGN.getTime();
	

	checkCudaErrors(hipFree(d_OutputIMGR));
	checkCudaErrors(hipFree(d_OutputIMGRx));
	checkCudaErrors(hipFree(d_OutputIMGRy));
	checkCudaErrors(hipFree(d_OutputIMGT));
	checkCudaErrors(hipFree(d_OutputBicubic));
	checkCudaErrors(hipFree(dInput_iU));
	checkCudaErrors(hipFree(dInput_iV));
	checkCudaErrors(hipFree(dInput_fPXY));
	checkCudaErrors(hipFree(dOutput_fDP));
	free(hInput_dR);
	free(hInput_dT);
	free(fZNCC);
	free(fdP);
	free(fdPXY);
	free(iU);
	free(iV);
}