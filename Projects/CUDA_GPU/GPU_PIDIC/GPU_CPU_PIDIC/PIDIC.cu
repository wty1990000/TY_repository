#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "EasyBMP.h"
#include "PIDIC.cuh"
#include "FFTCC.h"

#include <stdio.h>
#include <iostream>

const int BLOCK_SIZE = 16;

//CUDA RUNTIME Initialization
void InitCuda()
{
	hipFree(0);
}

__global__ void precomputation_kernel(float *d_InputIMGR, float *d_InputIMGT, const float* __restrict__ d_InputBiubicMatrix,
								 float *d_OutputIMGR, float *d_OutputIMGT, float *d_OutputIMGRx, float *d_OutputIMGRy,
								 float *d_OutputIMGTx, float *d_OutputIMGTy, float *d_OutputIMGTxy, float *d_OutputdtBicubic,
								 int width, int height)
{
	//The size of input images
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	//Temp arrays
	float d_TaoT[16];
	float d_AlphaT[16];

	//The rows and cols of output matrix.

	if((row < height) && (col < width)){
		d_OutputIMGR[row*width+col]  = d_InputIMGR[(row+1)*(width+2)+col+1];
		d_OutputIMGRx[row*width+col] = 0.5 * (d_InputIMGR[(row+1)*(width+2)+col+2] - d_InputIMGR[(row+1)*(width+2)+col]);
		d_OutputIMGRy[row*width+col] = 0.5 * (d_InputIMGR[(row+2)*(width+2)+col+1] - d_InputIMGR[(row)*(width+2)+col+1]);

		d_OutputIMGT[row*width+col]  = d_InputIMGT[(row+1)*(width+2)+col+1];
		d_OutputIMGTx[row*width+col] = 0.5 * (d_InputIMGT[(row+1)*(width+2)+col+2] -d_InputIMGT[(row+1)*(width+2)+col]);
		d_OutputIMGTy[row*width+col] = 0.5 * (d_InputIMGT[(row+2)*(width+2)+col+1] - d_InputIMGT[(row)*(width+2)+col+1]);
		d_OutputIMGTxy[row*width+col]= 0.25 * (d_InputIMGT[(row+2)*(width+2)+col+2]  - d_InputIMGT[(row)*(width+2)+col+2] -d_InputIMGT[(row+2)*(width+2)+col] + d_InputIMGT[(row)*(width+2)+col]);
	}
	__syncthreads();
	if((row < height-1) && (col < width-1)){
		d_TaoT[0] = d_OutputIMGT[row*(width)+col];
		d_TaoT[1] = d_OutputIMGT[row*(width)+col+1];
		d_TaoT[2] = d_OutputIMGT[(row+1)*(width)+col];
		d_TaoT[3] = d_OutputIMGT[(row+1)*(width)+col+1];
		d_TaoT[4] = d_OutputIMGTx[row*(width)+col];
		d_TaoT[5] = d_OutputIMGTx[row*(width)+col+1];
		d_TaoT[6] = d_OutputIMGTx[(row+1)*(width)+col];
		d_TaoT[7] = d_OutputIMGTx[(row+1)*(width)+col+1];
		d_TaoT[8] = d_OutputIMGTy[row*(width)+col];
		d_TaoT[9] = d_OutputIMGTy[row*(width)+col+1];
		d_TaoT[10] = d_OutputIMGTy[(row+1)*(width)+col];
		d_TaoT[11] = d_OutputIMGTy[(row+1)*(width)+col+1];
		d_TaoT[12] = d_OutputIMGTxy[row*(width)+col];
		d_TaoT[13] = d_OutputIMGTxy[row*(width)+col+1];
		d_TaoT[14] = d_OutputIMGTxy[(row+1)*(width)+col];
		d_TaoT[15] = d_OutputIMGTxy[(row+1)*(width)+col+1];
		for(int k=0; k<16; k++){
			d_AlphaT[k] = 0.0;
			for(int l=0; l<16; l++){
				d_AlphaT[k] += (d_InputBiubicMatrix[k*16+l] * d_TaoT[l]);
			}
		}
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+0] = d_AlphaT[0];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+1] = d_AlphaT[1];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+2] = d_AlphaT[2];
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+3] = d_AlphaT[3];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+0] = d_AlphaT[4];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+1] = d_AlphaT[5];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+2] = d_AlphaT[6];
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+3] = d_AlphaT[7];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+0] = d_AlphaT[8];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+1] = d_AlphaT[9];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+2] = d_AlphaT[10];
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+3] = d_AlphaT[11];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+0] = d_AlphaT[12];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+1] = d_AlphaT[13];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+2] = d_AlphaT[14];
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+3] = d_AlphaT[15];
	}
	else if(((row >=height-1)&&(row < height)) && ((col >= width-1)&&(col<width))){
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+0)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+1)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+2)*4+3] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+0] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+1] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+2] = 0.0;
		d_OutputdtBicubic[((row*(width)+col)*4+3)*4+3] = 0.0;
	}

}
